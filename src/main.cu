// src/main.cu
#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel to add two arrays
__global__ void add_arrays(int *a, int *b, int *result, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        result[index] = a[index] + b[index];
    }
}

int main() {
    int n = 1000; // Size of the arrays
    int *a, *b, *result;
    int *d_a, *d_b, *d_result;

    // Allocate host memory
    a = new int[n];
    b = new int[n];
    result = new int[n];

    // Initialize the arrays
    for (int i = 0; i < n; ++i) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Allocate device memory
    hipMalloc((void**)&d_a, n * sizeof(int));
    hipMalloc((void**)&d_b, n * sizeof(int));
    hipMalloc((void**)&d_result, n * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel with enough blocks and threads
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    add_arrays<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_result, n);

    // Copy result from device to host
    hipMemcpy(result, d_result, n * sizeof(int), hipMemcpyDeviceToHost);

    // Print a few results
    for (int i = 0; i < 10; ++i) {
        std::cout << "Result[" << i << "] = " << result[i] << std::endl;
    }

    // Free memory
    delete[] a;
    delete[] b;
    delete[] result;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    return 0;
}
